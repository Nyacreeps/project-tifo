#include "hip/hip_runtime.h"
#include "filter.cuh"

[[gnu::noinline]]
void _abortError(const char *msg, const char *fname, int line) {
    hipError_t err = hipGetLastError();
    spdlog::error("{} ({}, line: {})", msg, fname, line);
    spdlog::error("Error {}: {}", hipGetErrorName(err), hipGetErrorString(err));
    std::exit(1);
}

#define abortError(msg) _abortError(msg, __FUNCTION__, __LINE__)

template <int radius>
constexpr bool *circle_mask() {
    int size = 2 * radius + 1;
    auto result = (bool *)malloc(sizeof(bool) * size * size);
    for (int i = 0; i < size; i++)
        for (int j = 0; j < size; j++)
            result[i * size + j] = std::pow(i - radius, 2) + std::pow(j - radius, 2) <= std::pow(radius, 2);
    return result;
}

unsigned char *flatten(unsigned char **buffer_, const int width, const int height) {
    auto result = (unsigned char *)malloc(sizeof(unsigned char) * width * height);
#pragma omp parallel for schedule(dynamic) shared(buffer_, result, width, height) default(none) collapse(2) nowait
    for (int i = 0; i < height; i++)
        for (int j = 0; j < width; j++)
            result[i * width + j] = buffer_[i][j];
    return result;
}

unsigned char **unflatten(unsigned char *buffer_, const int width, const int height) {
    auto result = (unsigned char **)malloc(sizeof(unsigned char *) * height);
    for (int i = 0; i < height; result[i++] = (unsigned char *)malloc(sizeof(unsigned char) * width));
#pragma omp parallel for schedule(dynamic) shared(buffer_, result, width, height) default(none) collapse(2) nowait
    for (int i = 0; i < height; i++)
        for (int j = 0; j < width; j++)
            result[i][j] = buffer_[i * width + j];
    return result;
}

__global__ void compute_intensities(unsigned char *buffer, size_t bufferPitch, size_t width, size_t height,
                        unsigned char *res, size_t resPitch)
{
    const int x = blockDim.x * blockIdx.x + threadIdx.x;
    const int y = blockDim.y * blockIdx.y + threadIdx.y;

    if (x >= width || y >= height)
        return;

    const auto *buffer_line = buffer + y * bufferPitch;
    auto *res_line = res + y * resPitch;
    const auto r = buffer_line[x * 3];
    const auto g = buffer_line[x * 3 + 1];
    const auto b = buffer_line[x * 3 + 2];

    int intensity = (int)((double)(r + g + b) / 3 * INTENSITY / 255);
    res_line[x] = intensity < 0 ? 0 : intensity > 255 ? 255 : intensity;
}

template <int radius>
__global__ void get_Imax(unsigned char *I, size_t I_Pitch, size_t width, size_t height,
                             const bool *mask, size_t maskPitch, unsigned char *I_max, size_t I_maxPitch)
{
    const int x = blockDim.x * blockIdx.x + threadIdx.x;
    const int y = blockDim.y * blockIdx.y + threadIdx.y;

    if (x >= width || y >= height)
        return;

    unsigned char intensityBins[256];
    for (int i = 0; i < 256; intensityBins[i++] = 0);

    unsigned char max = 0;
    unsigned char imax = 0;
    for (int i = 0; i < 2 * radius + 1; ++i) {
        if (y + i - radius >= 0 && y + i - radius < height) {
            const auto *mask_line = (bool *) ((char *) mask + i * maskPitch);
            for (int j = 0; j < 2 * radius + 1; ++j) {
                const auto idxh = y + i - radius;
                const auto idxw = x + j - radius;
                if (mask_line[j] && idxw >= 0 && idxw < width) {
                    const auto intensity = (I + idxh * I_Pitch)[idxw];
                    auto tmp = ++intensityBins[intensity];
                    if (tmp > max) {
                        max = tmp;
                        imax = intensity;
                    }
                }
            }
        }
    }
    (I_max + y * I_maxPitch)[x] = imax;
}

template <int radius>
__global__ void apply_filter(unsigned char *buffer, size_t bufferPitch, size_t width, size_t height,
                             unsigned char *res, size_t resPitch, const bool *mask, size_t maskPitch,
                             unsigned char *I, size_t I_Pitch, unsigned char *I_max, size_t I_maxPitch)
{
    const int x = blockDim.x * blockIdx.x + threadIdx.x;
    const int y = blockDim.y * blockIdx.y + threadIdx.y;

    if (x >= width || y >= height)
        return;

    auto *res_line = (unsigned char *) ((char *) res + y * resPitch);
    const auto indmax = (I_max + y * I_maxPitch)[x];

    int R = 0, G = 0, B = 0;
    unsigned char n = 0;
    for (int i = 0; i < 2 * radius + 1; ++i) {
        if (y + i - radius >= 0 && y + i - radius < height) {
            const auto *buffer_line = buffer + (y + i - radius) * bufferPitch;
            const auto *mask_line = (bool *)((char *)mask + i * maskPitch);
            for (int j = 0; j < 2 * radius + 1; ++j) {
                const auto idxh = y + i - radius;
                const auto idxw = x + j - radius;
                if (mask_line[j] && idxw >= 0 && idxw < width) {
                    const auto intensity = (I + idxh * I_Pitch)[idxw];
                    if (intensity == indmax) {
                        const auto color = &(buffer_line[idxw * 3]);
                        R += (int) color[0];
                        G += (int) color[1];
                        B += (int) color[2];
                        ++n;
                    }
                }
            }
        }
    }

    res_line[x * 3] = (int)round((double)R / (double)n);
    res_line[x * 3 + 1] = (int)round((double)G / (double)n);
    res_line[x * 3 + 2] = (int)round((double)B / (double)n);
}

static unsigned char *oil_filter_(unsigned char *devBuffer, size_t *Pitch, const int width, const int height)
{
    int bsize = 32;
    int w = std::ceil((float) width / (float)bsize);
    int h = std::ceil((float) height / (float)bsize);

    spdlog::debug("running kernel of size ({},{})", w, h);

    dim3 dimBlock(bsize, bsize);
    dim3 dimGrid(w, h);

    const auto BufferPitch = *Pitch;

    // Device I
    unsigned char *devI;
    size_t IPitch;
    if (hipMallocPitch(&devI, &IPitch, width * sizeof(unsigned char), height) != hipSuccess)
        abortError("Fail Ix allocation");
    compute_intensities<<<dimGrid, dimBlock>>>(devBuffer, BufferPitch, width, height,
                                               devI, IPitch);
    hipDeviceSynchronize();

    // Device mask
    bool *devMask;
    size_t MaskPitch;
    if (hipMallocPitch(&devMask, &MaskPitch, (2 * RADIUS + 1) * sizeof(bool), 2 * RADIUS + 1) != hipSuccess)
        abortError("Fail mask allocation");
    const auto mask = circle_mask<RADIUS>();
    if (hipMemcpy2D(devMask, MaskPitch, mask, (2 * RADIUS + 1) * sizeof(bool),
                     (2 * RADIUS + 1) * sizeof(bool), (2 * RADIUS + 1), hipMemcpyHostToDevice) != hipSuccess)
        abortError("Fail mask copy");
    free(mask);

    // Device Imax
    unsigned char *devImax;
    size_t ImaxPitch;
    if (hipMallocPitch(&devImax, &ImaxPitch, width * sizeof(unsigned char), height) != hipSuccess)
        abortError("Fail Imax allocation");

    get_Imax<RADIUS><<<dimGrid, dimBlock>>>(devI, IPitch, width, height,
                                            devMask, MaskPitch,devImax, ImaxPitch);
    hipDeviceSynchronize();

    // Device result
    unsigned char *devRes;
    size_t ResPitch;
    if (hipMallocPitch(&devRes, &ResPitch, width * 3 * sizeof(unsigned char), height) != hipSuccess)
        abortError("Fail result allocation");

    apply_filter<RADIUS><<<dimGrid, dimBlock>>>(devBuffer, BufferPitch, width, height,
                                                devRes, ResPitch, devMask, MaskPitch,
                                                devI, IPitch, devImax, ImaxPitch);
    hipDeviceSynchronize();
    hipFree(devI);
    hipFree(devImax);
    hipFree(devMask);

    *Pitch = ResPitch;
    return devRes;
}


unsigned char *oil_filter(unsigned char *buffer, const int width, const int height)
{
    // Device image buffer
    unsigned char *devBuffer;
    size_t Pitch;
    if (hipMallocPitch(&devBuffer, &Pitch, width * 3 * sizeof(unsigned char), height) != hipSuccess)
        abortError("Fail buffer allocation");
    if (hipMemcpy2D(devBuffer, Pitch, buffer, width * 3 * sizeof(unsigned char),
                     width * 3 * sizeof(unsigned char), height, hipMemcpyHostToDevice) != hipSuccess)
        abortError("Fail buffer copy");

    auto res = (unsigned char *)malloc( height * width * 3 * sizeof(unsigned char));

    auto devRes = oil_filter_(devBuffer, &Pitch, width, height);
    hipFree(devBuffer);

    if (hipMemcpy2D(res, width * 3 * sizeof(unsigned char), devRes, Pitch,
                     width * 3 * sizeof(unsigned char), height, hipMemcpyDeviceToHost) != hipSuccess)
        abortError("Fail result copy");
    hipFree(devRes);

    return res;
}

__global__ void yuv2rgb(unsigned char *yuv, const int width, const int height, unsigned char *rgb, size_t pitch)
{
    const int x = blockDim.x * blockIdx.x + threadIdx.x;
    const int y = blockDim.y * blockIdx.y + threadIdx.y;

    if (x >= width || y >= height)
        return;

    const int size = width * height;
    const size_t Ubase = size;
    const size_t Vbase = size * 1.25;

    auto Y = yuv[y * width + x];
    auto U = yuv[Ubase + y * width / 4 + x / 2];
    auto V = yuv[Vbase + y * width / 4 + x / 2];

    double R = (double)Y + 1.140 * (double)V;
    double G = Y - 0.395 * (double)U - 0.581 * (double)V;
    double B = Y + 2.032 * (double)U;

    auto rgb_buffer = rgb + y * pitch;

    rgb_buffer[x * 3] = R > 255 ? 255 : R < 0 ? 0 : (unsigned char)R;
    rgb_buffer[x * 3 + 1] = (G > 255) ? 255 : G < 0 ? 0 : (unsigned char)G;
    rgb_buffer[x * 3 + 2] = (B > 255) ? 255 : B < 0 ? 0 : (unsigned char)B;
}

__global__ void rgb2yuv(unsigned char *rgb, size_t pitch, const int width, const int height, unsigned char *yuv)
{
    const int x = blockDim.x * blockIdx.x + threadIdx.x;
    const int y = blockDim.y * blockIdx.y + threadIdx.y;

    if (x >= width || y >= height)
        return;

    const int size = width * height;
    const size_t Ubase = size;
    const size_t Vbase = size * 1.25;

    const auto rgb_line = rgb + y * pitch;
    auto R = rgb_line[x * 3];
    auto G = rgb_line[x * 3 + 1];
    auto B = rgb_line[x * 3 + 2];

    double Y = 0.299 * R + 0.587 * G + 0.114 * B;
    double U = 0.492 * (B - Y);
    double V = 0.877 * (R - Y);

    yuv[y * width + x] = (Y > 255) ? 255 : Y < 0 ? 0 : (unsigned char)Y;
    yuv[Ubase + y * width / 4 + x / 2] = U > 255 ? 255 : U < 0 ? 0 : (unsigned char)U;
    yuv[Vbase + y * width / 4 + x / 2] = V > 255 ? 255 : V < 0 ? 0 : (unsigned char)V;
}

unsigned char *oil_filter_yuv420(unsigned char *buffer, const int width, const int height)
{
    int bsize = 32;
    int w = std::ceil((float) width / (float)bsize);
    int h = std::ceil((float) height / (float)bsize);

    dim3 dimBlock(bsize, bsize);
    dim3 dimGrid(w, h);

    // Device Yuvbuffer
    unsigned char *devYuvBuffer;
    if (hipMalloc(&devYuvBuffer, width * height * 3 / 2) != hipSuccess)
        abortError("Fail buffer allocation");
    if (hipMemcpy(devYuvBuffer, buffer, width * height * 3 / 2, hipMemcpyHostToDevice) != hipSuccess)
        abortError("Fail buffer copy");

    // Device image buffer
    unsigned char *devBuffer;
    size_t Pitch;
    if (hipMallocPitch(&devBuffer, &Pitch, width * 3 * sizeof(unsigned char), height) != hipSuccess)
        abortError("Fail buffer allocation");
    yuv2rgb<<<dimGrid, dimBlock>>>(devYuvBuffer, width, height, devBuffer, Pitch);
    hipDeviceSynchronize();

    auto devRes = oil_filter_(devBuffer, &Pitch, width, height);
    hipFree(devBuffer);

    rgb2yuv<<<dimGrid, dimBlock>>>(devRes, Pitch, width, height, devYuvBuffer);
    hipDeviceSynchronize();
    hipFree(devRes);

    auto res = (unsigned char *)malloc( height * width * 3 / 2);

    if (hipMemcpy(res, devYuvBuffer, width * height * 3 / 2, hipMemcpyDeviceToHost) != hipSuccess)
        abortError("Fail result copy");
    hipFree(devYuvBuffer);

    return res;
}