#include "hip/hip_runtime.h"
#include "filter.cuh"

[[gnu::noinline]]
void _abortError(const char *msg, const char *fname, int line) {
    hipError_t err = hipGetLastError();
    spdlog::error("{} ({}, line: {})", msg, fname, line);
    spdlog::error("Error {}: {}", hipGetErrorName(err), hipGetErrorString(err));
    std::exit(1);
}

#define abortError(msg) _abortError(msg, __FUNCTION__, __LINE__)

template <int radius>
constexpr bool *circle_mask() {
    int size = 2 * radius + 1;
    auto result = (bool *)malloc(sizeof(bool) * size * size);
    for (int i = 0; i < size; i++)
        for (int j = 0; j < size; j++)
            result[i * size + j] = std::pow(i - radius, 2) + std::pow(j - radius, 2) <= std::pow(radius, 2);
    return result;
}

unsigned char *flatten(unsigned char **buffer_, const int width, const int height) {
    auto result = (unsigned char *)malloc(sizeof(unsigned char) * width * height);
#pragma omp parallel for schedule(dynamic) shared(buffer_, result, width, height) default(none) collapse(2) nowait
    for (int i = 0; i < height; i++)
        for (int j = 0; j < width; j++)
            result[i * width + j] = buffer_[i][j];
    return result;
}

unsigned char **unflatten(unsigned char *buffer_, const int width, const int height) {
    auto result = (unsigned char **)malloc(sizeof(unsigned char *) * height);
    for (int i = 0; i < height; result[i++] = (unsigned char *)malloc(sizeof(unsigned char) * width));
#pragma omp parallel for schedule(dynamic) shared(buffer_, result, width, height) default(none) collapse(2) nowait
    for (int i = 0; i < height; i++)
        for (int j = 0; j < width; j++)
            result[i][j] = buffer_[i * width + j];
    return result;
}

__global__ void compute_intensities(unsigned char *buffer, size_t bufferPitch, size_t width, size_t height,
                        unsigned char *res, size_t resPitch)
{
    const int x = blockDim.x * blockIdx.x + threadIdx.x;
    const int y = blockDim.y * blockIdx.y + threadIdx.y;

    if (x >= width || y >= height)
        return;

    const auto *buffer_line = buffer + y * bufferPitch;
    auto *res_line = res + y * resPitch;
    const auto r = buffer_line[x * 3];
    const auto g = buffer_line[x * 3 + 1];
    const auto b = buffer_line[x * 3 + 2];

    int intensity = (int)((double)(r + g + b) / 3 * INTENSITY / 255);
    res_line[x] = intensity < 0 ? 0 : intensity > 255 ? 255 : intensity;
}

template <int radius>
__global__ void get_Imax(unsigned char *I, size_t I_Pitch, size_t width, size_t height,
                             const bool *mask, size_t maskPitch, unsigned char *I_max, size_t I_maxPitch)
{
    const int x = blockDim.x * blockIdx.x + threadIdx.x;
    const int y = blockDim.y * blockIdx.y + threadIdx.y;

    if (x >= width || y >= height)
        return;

    unsigned char intensityBins[256];
    for (int i = 0; i < 256; intensityBins[i++] = 0);

    unsigned char max = 0;
    unsigned char imax = 0;
    for (int i = 0; i < 2 * radius + 1; ++i) {
        if (y + i - radius >= 0 && y + i - radius < height) {
            const auto *mask_line = (bool *) ((char *) mask + i * maskPitch);
            for (int j = 0; j < 2 * radius + 1; ++j) {
                const auto idxh = y + i - radius;
                const auto idxw = x + j - radius;
                if (mask_line[j] && idxw >= 0 && idxw < width) {
                    const auto intensity = (I + idxh * I_Pitch)[idxw];
                    auto tmp = ++intensityBins[intensity];
                    if (tmp > max) {
                        max = tmp;
                        imax = intensity;
                    }
                }
            }
        }
    }
    (I_max + y * I_maxPitch)[x] = imax;
}

template <int radius>
__global__ void apply_filter(unsigned char *buffer, size_t bufferPitch, size_t width, size_t height,
                             unsigned char *res, size_t resPitch, const bool *mask, size_t maskPitch,
                             unsigned char *I, size_t I_Pitch, unsigned char *I_max, size_t I_maxPitch)
{
    const int x = blockDim.x * blockIdx.x + threadIdx.x;
    const int y = blockDim.y * blockIdx.y + threadIdx.y;

    if (x >= width || y >= height)
        return;

    auto *res_line = (unsigned char *) ((char *) res + y * resPitch);
    const auto indmax = (I_max + y * I_maxPitch)[x];

    int R = 0, G = 0, B = 0;
    unsigned char n = 0;
    for (int i = 0; i < 2 * radius + 1; ++i) {
        if (y + i - radius >= 0 && y + i - radius < height) {
            const auto *buffer_line = buffer + (y + i - radius) * bufferPitch;
            const auto *mask_line = (bool *)((char *)mask + i * maskPitch);
            for (int j = 0; j < 2 * radius + 1; ++j) {
                const auto idxh = y + i - radius;
                const auto idxw = x + j - radius;
                if (mask_line[j] && idxw >= 0 && idxw < width) {
                    const auto intensity = (I + idxh * I_Pitch)[idxw];
                    if (intensity == indmax) {
                        const auto color = &(buffer_line[idxw * 3]);
                        R += (int) color[0];
                        G += (int) color[1];
                        B += (int) color[2];
                        ++n;
                    }
                }
            }
        }
    }

    res_line[x * 3] = (int)round((double)R / (double)n);
    res_line[x * 3 + 1] = (int)round((double)G / (double)n);
    res_line[x * 3 + 2] = (int)round((double)B / (double)n);
}


unsigned char **oil_filter(unsigned char **buffer_, const int width, const int height)
{
    int bsize = 32;
    int w = std::ceil((float) width / (float)bsize);
    int h = std::ceil((float) height / (float)bsize);

    spdlog::debug("running kernel of size ({},{})", w, h);

    dim3 dimBlock(bsize, bsize);
    dim3 dimGrid(w, h);

    // Device image buffer
    unsigned char *devBuffer;
    size_t BufferPitch;
    if (hipMallocPitch(&devBuffer, &BufferPitch, width * 3 * sizeof(unsigned char), height) != hipSuccess)
        abortError("Fail buffer allocation");
    const auto buffer = flatten(buffer_, width * 3, height);
    if (hipMemcpy2D(devBuffer, BufferPitch, buffer, width * 3 * sizeof(unsigned char),
                     width * 3 * sizeof(unsigned char), height, hipMemcpyHostToDevice) != hipSuccess)
        abortError("Fail buffer copy");
    free(buffer);

    // Device I
    unsigned char *devI;
    size_t IPitch;
    if (hipMallocPitch(&devI, &IPitch, width * sizeof(unsigned char), height) != hipSuccess)
        abortError("Fail Ix allocation");
    compute_intensities<<<dimGrid, dimBlock>>>(devBuffer, BufferPitch, width, height,
                                               devI, IPitch);
    hipDeviceSynchronize();

    // Device result
    unsigned char *devRes;
    size_t ResPitch;
    if (hipMallocPitch(&devRes, &ResPitch, width * 3 * sizeof(unsigned char), height) != hipSuccess)
        abortError("Fail result allocation");

    // Device mask
    bool *devMask;
    size_t MaskPitch;
    if (hipMallocPitch(&devMask, &MaskPitch, (2 * RADIUS + 1) * sizeof(bool), 2 * RADIUS + 1) != hipSuccess)
        abortError("Fail mask allocation");
    const auto mask = circle_mask<RADIUS>();
    if (hipMemcpy2D(devMask, MaskPitch, mask, (2 * RADIUS + 1) * sizeof(bool),
                     (2 * RADIUS + 1) * sizeof(bool), (2 * RADIUS + 1), hipMemcpyHostToDevice) != hipSuccess)
        abortError("Fail mask copy");
    free(mask);

    // Device Imax
    unsigned char *devImax;
    size_t ImaxPitch;
    if (hipMallocPitch(&devImax, &ImaxPitch, width * sizeof(unsigned char), height) != hipSuccess)
        abortError("Fail Imax allocation");

    get_Imax<RADIUS><<<dimGrid, dimBlock>>>(devI, IPitch, width, height,
                                                devMask, MaskPitch,devImax, ImaxPitch);
    hipDeviceSynchronize();
    apply_filter<RADIUS><<<dimGrid, dimBlock>>>(devBuffer, BufferPitch, width, height,
                                                devRes, ResPitch, devMask, MaskPitch,
                                                devI, IPitch, devImax, ImaxPitch);
    hipDeviceSynchronize();
    hipFree(devI);
    hipFree(devMask);
    hipFree(devBuffer);

    auto res_ = (unsigned char *)malloc( height * width * 3 * sizeof(unsigned char));

    if (hipMemcpy2D(res_, width * 3 * sizeof(unsigned char), devRes, ResPitch,
                     width * 3 * sizeof(unsigned char), height, hipMemcpyDeviceToHost) != hipSuccess)
        abortError("Fail result copy");
    hipFree(devRes);

    auto res = unflatten(res_, width * 3, height);
    free(res_);

    return res;
}
